
    #include <hip/hip_runtime.h>
    #include <cstdlib>
    #include <cstdio>
    #include <omp.h>
    #include <hip/hip_runtime.h>

    using namespace std;

    #include "utils.cu"
    #include "kernelGPU_AC.cu"

    void cpu_sim(int n, int pasos, bool *tablero, bool *temp, int nt){
        int i,j,neighbour_live_cell;
        double t;
        omp_set_num_threads(nt);
        for(int p=0; p<pasos; ++p){
                    printf("[AC][CPU][%i]\n", p);
                    t = omp_get_wtime();

                    #pragma omp parallel for
                    for(i=0; i<n; i++){
                        for(j=0;j<n;j++){
                            neighbour_live_cell=count_live_neighbour_cell(tablero,n,i,j);
                            if(tablero[i*n+j] && (neighbour_live_cell==2 || neighbour_live_cell==3)){
                                temp[i*n+j]=1;
                            }
                            else if((!tablero[i*n+j]) && neighbour_live_cell==3){
                                temp[i*n+j]=1;
                            }
                            else{
                            temp[i*n+j]=0;
                            }
                        }
                    }
                
                    
                    std::swap(tablero,temp);
                    if(n<=128) printAC(n,tablero);
                    printf("done in %f[s]\n", omp_get_wtime() - t);
                    printf("Press enter to continue\n");
                    fflush(stdout);
                    getchar();
        }
    }

    void gpu_sim(int n,int pasos,bool *tablero,bool *temp, int nb, int GPUID){
        bool *board_d, *temp_d;
        float msecs;
        hipEvent_t start, stop;
        hipEventCreate(&start); hipEventCreate(&stop);
        hipSetDevice(GPUID);
        hipMalloc(&board_d,sizeof(bool)*n*n);
        hipMalloc(&temp_d,sizeof(bool)*n*n);
        hipMemcpy(board_d,tablero,sizeof(bool)*n*n, hipMemcpyHostToDevice);
        dim3 block(nb,nb);
        dim3 grid((n+block.x-1)/block.x, (n+block.y-1)/block.y);
        for(int p=0; p<pasos; p++){
            printf("[AC][GPU][%i]\n", p);
            hipEventRecord(start);
            GoLKernel<<<grid, block>>>(board_d, temp_d, n);
            hipDeviceSynchronize();    hipEventRecord(stop);  hipEventSynchronize(stop);
            hipEventElapsedTime(&msecs,start,stop);
            hipMemcpy(board_d,temp_d,sizeof(bool)*n*n,hipMemcpyDeviceToDevice);
            hipMemcpy(tablero,board_d,sizeof(bool)*n*n,hipMemcpyDeviceToHost);
            if(n<=128) printAC(n,tablero);
             printf("done in %f[s]\n", msecs/1000.0f);
            printf("Press enter to continue\n");
            fflush(stdout);
            getchar();
        }

    }

    int main(int argc, char **argv){
        if(argc != 8){
            fprintf(stderr, "Error. Debe ejecutarse como ./prog <gpu-id> <n> <seed> <pasos> <nt> <nb> <cpu-o-gpu>\n\n");
            exit(EXIT_FAILURE);
        }
        long GPUID       = atoi(argv[1]);
        int n      = atoi(argv[2]);
        int seed     = atoi(argv[3]);
        int pasos    = atoi(argv[4]);
        int nt   = atoi(argv[5]);
        int nb  = atoi(argv[6]);
        int CPUoGPU = atoi(argv[7]);

        srand(seed);
        
        bool *tablero = (bool*)malloc(n*n*sizeof(bool));
        bool *temp = (bool*)malloc(n*n*sizeof(bool));
        for(int i=0; i<n; i++){
            for(int j=0; j<n; j++)
                tablero[i*n+j] = rand()%2;
        }
        printf("[AC][ORIGINAL]\n");
        printAC(n,tablero);
        printf("Press enter to continue\n");
        fflush(stdout);
        getchar();
        if(CPUoGPU){
            //modo GPU
            gpu_sim(n,pasos,tablero,temp,nb,GPUID);
        }
        else{
            //modo CPU
        cpu_sim(n,pasos,tablero,temp, nt);
        }
    }