#pragma once
#include <hip/hip_runtime.h>

__global__ void GoLKernel(bool *tablero, bool *temp, int n) {
    int fila = blockIdx.y * blockDim.y + threadIdx.y;
    int columna = blockIdx.x * blockDim.x + threadIdx.x;

    if (fila < 0 || fila >= n || columna < 0 || columna >= n) {
        return;
    }

    int cont = 0;
    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            if (i == 0 && j == 0) {
                continue;
            }

            int vecindad_eje_X = fila + i;
            int vecindad_eje_y = columna + j;
            if (vecindad_eje_X < 0 || vecindad_eje_X >= n || vecindad_eje_y < 0 || vecindad_eje_y >= n) {
                continue;
            }

            if (tablero[(vecindad_eje_X * n) + vecindad_eje_y])
                cont++;
        }
    }

    if (tablero[(fila * n) + columna]) {
        if (cont == 2 || cont == 3) {
            temp[(fila * n) + columna] = 1;
        } else {
            temp[(fila * n) + columna] = 0;
        }
    } else {
        if (cont == 3) {
            temp[(fila * n) + columna] = 1;
        } else {
            temp[(fila * n) + columna] = 0;
        }
    }
}