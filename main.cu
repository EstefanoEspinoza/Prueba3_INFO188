    #include <hip/hip_runtime.h>
    #include <cstdlib>
    #include <cstdio>
    #include <omp.h>
    #include <hip/hip_runtime.h>

    using namespace std;

    #include "utils.cu"
    #include "kernelGPU_AC.cu"

    void cpu_sim(int n, int pasos, bool *board, bool *temp, int nt){
        int neighbour_live_cell;
        double t,f;
        omp_set_num_threads(nt);
        for(int p = 0; p < pasos; p++){
                    printf("[AC][CPU] paso %i, n %i, nt %i\n", p,n,nt);
                    t = omp_get_wtime();

                    #pragma omp parallel for
                    for(int i = 0 ; i < n; i++){
                        for(int j = 0; j < n; j++){
                            neighbour_live_cell=count_live_neighbour_cell(board,n,i,j);
                            if(board[(i * n) + j] && (neighbour_live_cell == 2 || neighbour_live_cell == 3)){
                                temp[(i * n) + j] = 1;
                            }
                            else if((!board[(i * n) + j]) && neighbour_live_cell == 3){
                                temp[(i * n) + j] = 1;
                            }
                            else{
                            temp[(i * n) + j] = 0;
                            }
                        }
                    }
                
                    f = omp_get_wtime() - t;
                    std::swap(board,temp);
                    if(n<=128){
                        printAutomataCelular(n,board);
                    }
                    printf("terminado en %f[s]\n", f);
                    printf("Presiona enter para continuar\n");
                    fflush(stdout);
                    getchar();
        }
    }

    void gpu_sim(int n,int pasos,bool *board,bool *temp, int nb, int GPUID){
        bool *board_d, *temp_d;
        float msecs;
        hipEvent_t start, stop;
        hipEventCreate(&start); hipEventCreate(&stop);
        hipSetDevice(GPUID);
        hipMalloc(&board_d,sizeof(bool)*n*n);
        hipMalloc(&temp_d,sizeof(bool)*n*n);
        hipMemcpy(board_d,board,sizeof(bool)*n*n, hipMemcpyHostToDevice);
        dim3 block(nb,nb);
        dim3 grid((n + block.x - 1)/block.x, (n + block.y - 1)/block.y);
        for(int p = 0; p < pasos; p++){
            printf("[AC][GPU] paso %i, n %i, nb %i\n", p,n,nb);
            hipEventRecord(start);
            GoLKernel<<<grid, block>>>(board_d, temp_d, n);
            hipDeviceSynchronize();    hipEventRecord(stop);  hipEventSynchronize(stop);
            hipEventElapsedTime(&msecs,start,stop);
            hipMemcpy(board_d,temp_d,sizeof(bool)*n*n,hipMemcpyDeviceToDevice);
            if(n<=128){
                hipMemcpy(board,board_d,sizeof(bool)*n*n,hipMemcpyDeviceToHost);
                printAutomataCelular(n,board);
            }
            printf("terminado en %f[s]\n", msecs/1000.0f);
            printf("Presione enter para continuar\n");
            fflush(stdout);
            getchar();
        }
        if(n>128){
            hipMemcpy(board,board_d,sizeof(bool)*n*n,hipMemcpyDeviceToHost);
            }
    }

    int main(int argc, char **argv){
        if(argc != 8){
            fprintf(stderr, "Error. Debe ejecutarse como ./prog Id_GPU(0,1,2) n semilla pasos Numero_Threads Numero_Bloques cpu = 0 o gpu = 1 \n\n");
            exit(EXIT_FAILURE);
        }
        long GPUID       = atoi(argv[1]);
        int n      = atoi(argv[2]);
        int seed     = atoi(argv[3]);
        int pasos    = atoi(argv[4]);
        int nt   = atoi(argv[5]);
        int nb  = atoi(argv[6]); // si el valor ingresado es 0 lanza core dumped xd
        int CPUoGPU = atoi(argv[7]);

        srand(seed);
        
        bool *board = (bool*)malloc(n*n*sizeof(bool));
        bool *temp = (bool*)malloc(n*n*sizeof(bool));
        for(int i = 0; i < n; i++){
            for(int j = 0; j < n; j++)
                board[(i * n) + j] = rand()%2;
        }
        
        printf("[AC][ORIGINAL] created\n");
        if(n<=128){
            printAutomataCelular(n,board);
        } 
        printf("Presiona enter para continuar\n");
        fflush(stdout);
        getchar();
        if(CPUoGPU){
            //modo GPU
            gpu_sim(n,pasos,board,temp,nb,GPUID);
        }
        else{
            //modo CPU
        cpu_sim(n,pasos,board,temp, nt);
        }
    }
