#include "hip/hip_runtime.h"
    #pragma once
    
    #include <hip/hip_runtime.h>
    #include <cstdlib>
    #include <cstdio>
    #include <omp.h>
    #include <hip/hip_runtime.h>

    using namespace std;

    //#include "utils.cu"
    //#include "kernelGPU_AC.cu"

    // -----------------------------------------------------------------------------------------------

    #pragma once
__global__ void GoLKernel(bool *tablero, bool *temp, int n) {
    int fila = blockIdx.y * blockDim.y + threadIdx.y;
    int columna = blockIdx.x * blockDim.x + threadIdx.x;

    if (fila < 0 || fila >= n || columna < 0 || columna >= n) {
        return;
    }

    int cont = 0;
    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            if (i == 0 && j == 0) {
                continue;
            }

            int vecindad_eje_X = fila + i;
            int vecindad_eje_y = columna + j;
            if (vecindad_eje_X < 0 || vecindad_eje_X >= n || vecindad_eje_y < 0 || vecindad_eje_y >= n) {
                continue;
            }

            if (tablero[(vecindad_eje_X * n) + vecindad_eje_y])
                cont++;
        }
    }

    if (tablero[(fila * n) + columna]) {
        if (cont == 2 || cont == 3) {
            temp[(fila * n) + columna] = 1;
        } else {
            temp[(fila * n) + columna] = 0;
        }
    } else {
        if (cont == 3) {
            temp[(fila * n) + columna] = 1;
        } else {
            temp[(fila * n) + columna] = 0;
        }
    }
}

    // -----------------------------------------------------------------------------------------------

    void printAutomataCelular(int n,bool *board){
     for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++) {
            printf(board[i*n+j] ? "*" : "/");
        }
        printf("\n");
    }
}


int count_live_neighbour_cell(bool *board,int n, int r, int c){
    int i, j, count=0;
    for(i = r - 1; i < = r + 1; i++){
        for(j = c - 1; j < = c + 1; j++){
            if((i==r && j==c) || (i<0 || j<0) || (i>=n || j>=n)){
                continue;
            }
            if(board[i*n+j]){
                count++;
            }
        }
    }
    return count;
}

    // -----------------------------------------------------------------------------------------------

    void cpu_sim(int n, int pasos, bool *board, bool *temp, int nt){
        int neighbour_live_cell;
        double t,f;
        omp_set_num_threads(nt);
        for(int p = 0; p < pasos; p++){
                    printf("[AC][CPU] paso %i, n %i, nt %i\n", p,n,nt);
                    t = omp_get_wtime();

                    #pragma omp parallel for
                    for(int i = 0 ; i < n; i++){
                        for(int j = 0; j < n; j++){
                            neighbour_live_cell=count_live_neighbour_cell(board,n,i,j);
                            if(board[(i * n) + j] && (neighbour_live_cell == 2 || neighbour_live_cell == 3)){
                                temp[(i * n) + j] = 1;
                            }
                            else if((!board[(i * n) + j]) && neighbour_live_cell == 3){
                                temp[(i * n) + j] = 1;
                            }
                            else{
                            temp[(i * n) + j] = 0;
                            }
                        }
                    }
                
                    f = omp_get_wtime() - t;
                    std::swap(board,temp);
                    if(n<=128){
                        printAutomataCelular(n,board);
                    }
                    printf("terminado en %f[s]\n", f);
                    printf("Presiona enter para continuar\n");
                    fflush(stdout);
                    getchar();
        }
    }

    void gpu_sim(int n,int pasos,bool *board,bool *temp, int nb, int GPUID){
        bool *board_d, *temp_d;
        float msecs;
        hipEvent_t start, stop;
        hipEventCreate(&start); hipEventCreate(&stop);
        hipSetDevice(GPUID);
        hipMalloc(&board_d,sizeof(bool)*n*n);
        hipMalloc(&temp_d,sizeof(bool)*n*n);
        hipMemcpy(board_d,board,sizeof(bool)*n*n, hipMemcpyHostToDevice);
        dim3 block(nb,nb);
        dim3 grid((n + block.x - 1)/block.x, (n + block.y - 1)/block.y);
        for(int p = 0; p < pasos; p++){
            printf("[AC][GPU] paso %i, n %i, nb %i\n", p,n,nb);
            hipEventRecord(start);
            GoLKernel<<<grid, block>>>(board_d, temp_d, n);
            hipDeviceSynchronize();    hipEventRecord(stop);  hipEventSynchronize(stop);
            hipEventElapsedTime(&msecs,start,stop);
            hipMemcpy(board_d,temp_d,sizeof(bool)*n*n,hipMemcpyDeviceToDevice);
            if(n<=128){
                hipMemcpy(board,board_d,sizeof(bool)*n*n,hipMemcpyDeviceToHost);
                printAutomataCelular(n,board);
            }
            printf("terminado en %f[s]\n", msecs/1000.0f);
            printf("Presione enter para continuar\n");
            fflush(stdout);
            getchar();
        }
        if(n>128){
            hipMemcpy(board,board_d,sizeof(bool)*n*n,hipMemcpyDeviceToHost);
            }
    }

    int main(int argc, char **argv){
        if(argc != 8){
            fprintf(stderr, "Error. Debe ejecutarse como ./prog Id_GPU(0,1,2) n semilla pasos Numero_Threads Numero_Bloques cpu = 0 o gpu = 1 \n\n");
            exit(EXIT_FAILURE);
        }
        long GPUID       = atoi(argv[1]);
        int n      = atoi(argv[2]);
        int seed     = atoi(argv[3]);
        int pasos    = atoi(argv[4]);
        int nt   = atoi(argv[5]);
        int nb  = atoi(argv[6]); // si el valor ingresado en nb es 0 lanza core dumped xd
        int CPU_o_GPU = atoi(argv[7]);

        srand(seed);
        
        bool *board = (bool*)malloc(n*n*sizeof(bool));
        bool *temp = (bool*)malloc(n*n*sizeof(bool));
        for(int i = 0; i < n; i++){
            for(int j = 0; j < n; j++)
                board[(i * n) + j] = rand()%2;
        }
        
        printf("[AC][ORIGINAL] created\n");
        if(n<=128){
            printAutomataCelular(n,board);  //Imprime la primera iteración de la ejecución.
        } 
        printf("Presiona enter para continuar\n");
        fflush(stdout);
        getchar();
        if(CPU_o_GPU){
            //entra al modo GPU para esta ejecución
            gpu_sim(n,pasos,board,temp,nb,GPUID);
        }
        else{
            //entra al modo CPU para esta ejecución
        cpu_sim(n,pasos,board,temp, nt);
        }
    }
